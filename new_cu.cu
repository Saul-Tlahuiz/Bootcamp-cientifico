#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 2048

__global__void hola(void){
    register idx = threadIdx.x * blockDim.x * blockIdx.x + threadIdx.y + blockDim.y*blockIdx.y;
    __share__ A[2][2];
    __share__ B[2][2];
    if(blockIdx.x == 0 && blockIdx.y == 0)
    printf("Thread del primer grupo: %d, %d \n", threadIdx.x,threadIdx.y);
}

int main(){
    dim3 hilos(16,16)
    dim3 bloques(N/hilos.x,N/hilos.y);
    printf("%lu\n", sizeof(hilos));
    printf("%lu\n", sizeof(bloques));
    printf("%d, %d\n", hilos.x, hilos.y);
    printf("%d, %d\n", bloques.x, bloques.y);
    hola<<<bloques,hilos>>>();
    return 0;
}
