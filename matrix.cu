#include "hip/hip_runtime.h"
#include <stdlib>
#include <iostream>
#define N 256

__global__void matrizE(float A[N][N], float esc){
    int i = threadIdx.x;
    int j = threadIdx.y;
    A[i][j] = esc * A[i][j];
}

int main(void){

    float *matrizt;

    dim3 hilosBloque(N,N);

    matrizE<<<1,hilosBloque>>>(matrizt,4.5);

    return EXIT_SUCCESS;
}
