#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib>

#define N 2048

__global__void iniv(float *vector, int tam){
    int i = threadIdx.x + blockIdx.x * blockDim.x
    int brinco = blockDim.x*gridDim.x; //nos dice cuantos threads por bloque hay y cuantos bloques

    int y;
    if( i < tam){
        for(y = i; i < tam; y = y + brinco)
        vector[y] = 1.0;//podemos meter un for dentro de un kernel 
    }
}

__global__void sumaVect(float* vectA,float* vectB,float* vectC,int tam){
    int indice = threadIdx.x + blockId.x * blockDim.x;
    int brinco = blockDim.x*gridDim.x;
    int i;
    if(indice < tam){
        for(i = indice; i < tam; i = i*brinco)
        vectC[i] = vectA[i] + vectB[i];
    }
}

int main(void){
    float *vecta, *vectb, *salida;

    //int gpu = -1;
    //hipGetDeviceCount(&gpu); // esto se usa para acelerar las lecturas en la gpu
    int numthreads = 256;
    int numBloques = (N+ numthreads-1)/numthreads;
    std::cout<<numBloques<<' \n';

    hipMallocManaged(&vecta, N*sizeof(float));
    hipMallocManaged(&vectb, N*sizeof(float));
    hipMallocManaged(&salida, N*sizeof(float));
    
    iniv<<<2,256>>>(vecta,N);
    iniv<<<4,256>>>(vectb,N);
    iniv<<<numBloques, numthreads>>>(vecta, N);
    iniv<<<numBloques, numthreads>>>(vectb, N);
    hipDeviceSynchronize(); //lo necesitamos por que despues vamos a llamar un kernel

    sumaVect<<<4,256>>>(vecta,vectb,salida,N);

    for (int i = 0; i<N; i++)
    std::cout<<salida[i]<<' ';
    //std::cout<<vecta[i]<<std::' ';
    //std::cout<<vecta[512]<<std::endl;

    hipFree(vecta), hipFree(vectb), hipFree(salida);
    return EXIT_SUCCESS;
}