#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib>
#include <stdio.h>

#define N 1024

__device__ float primerSegmento(int indice, float sinuso){
    sinuso = sinuso+2.3;
    if(indice == 0)
    return(float) threadIdx.x;
    else
    return -1;
}

__global__void vecinos(float* vectorin, float* vectorout, int tam){
    register int i = threadIdx.x + blockDim.x * blockIdx.x;
    register float relativo = 0;

    if(i<tam)
    relativo = primerSegmento(i, vectorin[i]);

    if(i>0 && i<tam-1)
    vectorout[i] = vectorin[i-1] + vectorin[i] + vectorin[i+1] + relativo;
    else if(i==0 || i == (tam-1))
    vectorout[i] = vectorin[i];
}

int main(int argc, char** argv){
    float* fuente , *destino;
    hipError_t v_err;

    if(argc != 1)
    printf("%s\n", argv[1]);
    else
    exit(EXIT_FAILURE);


    hipMallocManaged(&fuente, N*sizeof(float));
    hipMallocManaged(&destino,N*sizeof(float));

    for(int i = 0; i<N; i++)
    fuente[i] = i;

    vecinos<<<1,N>>>(fuente, destino, N);
    hipDeviceSynchronize();
    v_err = hipGetLastError
    if(v_err != cudaSucces);
    return EXIT_SUCCES
}