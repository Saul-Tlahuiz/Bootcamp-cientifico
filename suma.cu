#include <stdio.h>
#include <hip/hip_runtime.h>

__globlal__ void escalav(float* esc, float* vect, int tam){
    vect[threadIdx.x] = *esc * vect[threadIdx.x];
}

int main(void){
    float a[10] = {1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0,1.0};
    float esc = 4.5;

    int tamv = 10;
    int tamd = size(float);

    float* d_a;
    float* d_esc;
    int* d_tamv;

    hipMalloc((void **)&d_a, tamv*tamv);
    hipMalloc((void **)&d_esc, tamd);

    hipMemcpy(d_a, &a, tamv*tamd, hipMemcpyHostToDevice);
    hipMemcpy(d_esc, &esc, tamd, hipMemcpyHostToDevice);

    escalav<<<1,10>>>(d_esc,d_a);

    hipMemcpy(a, d_a, tamv*tamd,hipMemcpyDeviceToHost);

    hipFree(d_a), hipFree(d_esc);
    return 0;
}