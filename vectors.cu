#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define TAMV 1024

__global__void inicializa(float* vect, int tam){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < tam)
    vect[i] = i;
}

__global__void escalav(float* esc, float *vect, int tam){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < tam)
    vect[i] = esc[i] + vect[i];
}

void inicializaH(float* vect, int tam){
    double* inutil;

    hipMallocManaged(&inutil, sizeof(double));
    inicializa<<<1, TAMV>>>(vect, tam);

    hipFree(inutil);
    return;
}

int main(void){
    float* vect;
    float* esc;

    hipError_t controlError;

    hipMallocManaged(&vector1, TAMV*sizeof(float));
    hipMallocManaged(&esc,sizeof(float));

    inicializaH(vector1,TAMV);
    hipDeviceSynchronize();
    controlError = hipGetLastError();
    if(controlError != cudaSucces){
        std::cout<<"Falla:"<<hipGetErrorString(controlError)<<std::endl;
        exit(EXIT_FAILURE);
    }

    *esc = 4.5;

    escalav<<<1,TAMV>>>(esc, vector1,TAMV);
    hipDeviceSynchronize();
    controlError = hipGetLastError();
    if(controlError != cudaSucces){
        std::cout<<<
    }
}