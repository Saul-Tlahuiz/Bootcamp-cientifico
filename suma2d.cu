#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 8

__global__void sumaMat(float A[N][N], float B[N][N], float C[N][N]){
    int i = threadIdx.x;
    int j = threadIdx.y;

    C[i][j] = A[i][j] + B[i][j];
}

int main(void){
    int bloques = 1;
    //dim3 es un dato unico de cuda y nos permite definir una variable ademas los kernel consumen este tipo de dato
    int N = 8;
    dim3 hilosporbloque(bloques, N, N);

    //definimos la matriz
    float Ma[N][N],Mb[N][N],Mc[N][N];
    
    sumaMat<<<1,(8,8)>>>

    return 0;
}