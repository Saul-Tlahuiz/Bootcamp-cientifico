
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void muln(float *a, float *b, float *c){
    *c = (*a) * (*b);
}

int main(void){
    float a,b,c; //host

    float *d_a, *d_b, *d_c; //device

    int size = sizeof(float);
    //reservamos memoria
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);
    a = 2.0;
    b = 5.2;

    //copiamos al device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    //lanzamos el kernel
    muln<<<1,1>>>(d_a,d_b,d_c);
    //recuperamos del device
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    //limpiamos la memoria en el device una por una
    hipFree(d_a);hipFree(d_b);hipFree(d_c);

    printf("%f\n",c);

    return 0;
}